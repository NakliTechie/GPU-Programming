// File: histogram_privatized_minimal_atomics.cu
// Description: A histogram implementation using privatization with minimal atomic usage.
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <cmath>

#define NUM_BINS 26 // For letters 'a' through 'z'

void checkCudaError(hipError_t err, const char* message) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << message << " (" << hipGetErrorString(err) << ")" << std::endl;
        exit(EXIT_FAILURE);
    }
}

__global__ void histogram_kernel(const char* data, int* global_bins, int N) {
    // 1. Declare private histogram in shared memory
    __shared__ int private_bins[NUM_BINS];
    __shared__ bool is_initialized;

    // 2. Initialize private histogram to zero - only one thread does this to prevent race conditions
    if (threadIdx.x == 0) {
        is_initialized = false;
    }
    __syncthreads(); // Wait for the flag to be set

    // Only initialize once all threads have reached this point
    if (threadIdx.x < NUM_BINS) {
        private_bins[threadIdx.x] = 0;
    }
    __syncthreads(); // Wait for all bins to be zeroed
    
    if (threadIdx.x == 0) {
        is_initialized = true;
    }
    __syncthreads(); // Wait for initialization to be marked as complete

    // 3. Each thread processes its data element
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N && is_initialized) {  // Only process after initialization is confirmed
        char value = data[idx];
        // Assuming input is lowercase 'a' through 'z'
        if (value >= 'a' && value <= 'z') {
            // Use a simple increment instead of atomic, since only one thread accesses each value
            // But still need atomic for the shared memory access to be safe
            atomicAdd(&private_bins[value - 'a'], 1);
        }
    }

    // 4. Wait for all threads to finish their private updates
    __syncthreads();

    // 5. Cooperatively add private results to the global bins
    if (threadIdx.x < NUM_BINS) {
        atomicAdd(&global_bins[threadIdx.x], private_bins[threadIdx.x]);
    }
}

int main() {
    int N = 8192; // Smaller problem size for better compatibility
    size_t data_size = N * sizeof(char);
    size_t bins_size = NUM_BINS * sizeof(int);

    std::vector<char> h_data(N);
    std::vector<int> h_bins_gpu(NUM_BINS, 0);
    std::vector<int> h_bins_cpu(NUM_BINS, 0);

    // Initialize data with a predictable pattern: 'a', 'b', 'c', ...
    for (int i = 0; i < N; ++i) {
        h_data[i] = 'a' + (i % NUM_BINS);
    }

    char* d_data;
    int* d_bins;
    hipMalloc(&d_data, data_size);
    hipMalloc(&d_bins, bins_size);

    hipMemcpy(d_data, h_data.data(), data_size, hipMemcpyHostToDevice);
    // Important: Initialize global bins to zero before kernel launch!
    hipMemset(d_bins, 0, bins_size);

    int threadsPerBlock = 256;
    int numBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    // Limit blocks to reduce resource usage
    if (numBlocks > 64) numBlocks = 64;

    std::cout << "Launching privatized histogram kernel with N=" << N << ", " << numBlocks << " blocks..." << std::endl;
    histogram_kernel<<<numBlocks, threadsPerBlock>>>(d_data, d_bins, N);
    hipDeviceSynchronize();
    std::cout << "Kernel finished." << std::endl;

    hipMemcpy(h_bins_gpu.data(), d_bins, bins_size, hipMemcpyDeviceToHost);

    // Verification on CPU
    for (int i = 0; i < N; ++i) {
        char value = h_data[i];
        if (value >= 'a' && value <= 'z') {
            h_bins_cpu[value - 'a']++;
        }
    }

    std::cout << "\n--- Sample Results (First 4 bins) ---" << std::endl;
    std::cout << "Bin | GPU Result | CPU Result" << std::endl;
    std::cout << "---------------------------------" << std::endl;
    for (int i = 0; i < 4; ++i) {
        std::cout << (char)('a' + i) << "   | " << h_bins_gpu[i] << "\t | " << h_bins_cpu[i] << std::endl;
    }

    bool success = true;
    for (int i = 0; i < NUM_BINS; ++i) {
        if (h_bins_gpu[i] != h_bins_cpu[i]) {
            success = false;
            break;
        }
    }
    std::cout << (success ? "\nVerification Successful!" : "\nVerification FAILED!") << std::endl;

    hipFree(d_data);
    hipFree(d_bins);

    return 0;
}